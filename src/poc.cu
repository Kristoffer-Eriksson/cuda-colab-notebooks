#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <fstream>
#include <iostream>
#include "stb_image_write.h"

__global__ void dummy_rgb_data(unsigned char *rgb_data, size_t width, size_t height) {
    for(int i = 0; i < width * height; i++){
        rgb_data[3 * i + 0] = 255;
        rgb_data[3 * i + 1] = 0;
        rgb_data[3 * i + 2] = 0;
    }
}

int main(int argc, char *argv[])
{
    if (argc != 2) {
        std::cout << "Usage: poc.exe <output_path.png>" << std::endl;
        return EXIT_FAILURE;
    }

    size_t width = 800;
    size_t height = 600;

    /** Generate image **/

    hipError_t error;

    unsigned char *rgb_data;

    /* Allocate n floats on device */
    error = hipMalloc((void **)&rgb_data, width * height * 3 * sizeof(unsigned char));

    if (error != hipSuccess) {
        std::cerr << "Error " << __FILE__ << ":" << __LINE__ << " error = " << error << std::endl;
        return EXIT_FAILURE;
    }

    dummy_rgb_data<<<1,1>>>(rgb_data, width, height);

    /** Transfer data from GPU to CPU **/

    unsigned char *host_rgb_data = (unsigned char *)malloc(width * height * 3 * sizeof(unsigned char));

    error = hipMemcpy(host_rgb_data, rgb_data, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);

    if (error != hipSuccess) {
        std::cerr << "Error " << __FILE__ << ":" << __LINE__ << " error = " << error << std::endl;
        return EXIT_FAILURE;
    }

    /** Save image as PNG **/

    stbi_write_png(argv[1], width, height, 3, host_rgb_data, width * 3);

    /** Cleanup **/

    hipFree(rgb_data);
    free(host_rgb_data);

    return EXIT_SUCCESS;
}
